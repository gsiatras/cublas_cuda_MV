// System Includes
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

// Cuda Includes
#include <hipblas.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>

// auxiliary functions
#include "AuxFuncs.h"

// Macro to store elements in a linear space in row-major format
#define IDX2C(i, j, ld) (((i) * (ld)) + (j))


int main(int argc, char ** argv) {
	int M, N = 0;
	// init the seed with current local time
	srand(time(NULL));

	// Get M - N values from arguments
	if (argc == 3){
		M = atoi(argv[1]);
		N = atoi(argv[2]);
	}
	else {
		fprintf(stderr, "Insufficient command line arguments!\n");
		fprintf(stderr, "USAGE: main <matrixHeight> <matrixWidth>\n");
		exit(-1);
	}

	hipblasStatus_t status;
	double * h_A, * h_b, * h_c; // host copies of a, b, c
	double * d_A, * d_b, * d_c; // device copies of a, b, c
	d_A = d_b = d_c = 0;

	const double alf = 1, bet = 0;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Initialize CUBLAS
	fprintf(stdout, "Using hipblasDgemv() test running..\n");
	status = cublasInit();
	if (status != HIPBLAS_STATUS_SUCCESS) {
		_error_handler("CUBLAS initialization error\n");
	}

	// Allocate host memory for the matrices
	((h_A = (double *) malloc(M * N * sizeof(double))) != 0) ?
	((h_b = (double *) malloc(N * sizeof(double))) != 0) ?
	((h_c = (double *) malloc(M * sizeof(double))) != 0) ?
	:
	_error_handler("host memory allocation error (C)\n") :
	_error_handler("host memory allocation error (B)\n") :
	_error_handler("host memory allocation error (A)\n") ;

	// Allocate device memory for the matrices
	((status = cublasAlloc(M * N, sizeof(double), (void**)&d_A)) == HIPBLAS_STATUS_SUCCESS) ?
	((status = cublasAlloc(N, sizeof(double), (void**)&d_b)) == HIPBLAS_STATUS_SUCCESS) ?
	((status = cublasAlloc(M, sizeof(double), (void**)&d_c)) == HIPBLAS_STATUS_SUCCESS) ?
	:
	_error_handler("device memory allocation error (C)\n") :
	_error_handler("device memory allocation error (B)\n") :
	_error_handler("device memory allocation error (A)\n") ;
	
	// Initialize matrix A and vector b with some values and also zero-ize c vector
	for (int i = 0; i < M; i++) {
		for (int j = 0; j < N; j++) {
			h_A[i*N + j] = randDouble();
		}
	}

	for (int i = 0; i < N; i++) {
		h_b[i] = randDouble();
	}

	for (int i = 0; i < M; i++) {
		h_c[i] = 0;
	}

	// Initialize the device matrices with the host matrices
	((status = hipblasSetMatrix(M, N, sizeof(double), h_A, M, d_A, M)) == HIPBLAS_STATUS_SUCCESS) ?
	((status = hipblasSetVector(N, sizeof(double), h_b, 1, d_b, 1)) == HIPBLAS_STATUS_SUCCESS) ?
	((status = hipblasSetVector(M, sizeof(double), h_c, 1, d_c, 1)) == HIPBLAS_STATUS_SUCCESS) ?
	:
	_error_handler("device access error (write C)\n") :
	_error_handler("device access error (write B)\n") :
	_error_handler("device access error (write A)\n") ;

	// Create a handle for CUBLAS
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	
	// Performs operation using CUBLAS
	hipEventRecord(start);
	hipblasDgemv(handle, HIPBLAS_OP_T, N, M, &alf, d_A, N, d_b, 1, &bet, d_c, 1);
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	fprintf(stdout, "Execution completed. Elapsed Time = %6.8f msecs\n", milliseconds);

	// Destroy the handle
	hipblasDestroy(handle);

	if ((status = cublasGetError()) != HIPBLAS_STATUS_SUCCESS) {
		_error_handler("kernel execution error.\n");
		return EXIT_FAILURE;
	}

	// Read the result back
	status = hipblasGetVector(M, sizeof(h_c[0]), d_c, 1, h_c, 1);
	if (status != HIPBLAS_STATUS_SUCCESS) {
		_error_handler("device access error (read C)\n");
	}

	fprintf(stdout, "Result: \n");
	for (int i = 0; i < M; i++) {
		fprintf(stdout, "%6.8f ", h_c[i]);
	}
	fprintf(stdout, "\n");

	/*fprintf(stdout, "\n A: ");
	for (int i = 0; i < 2*N; i++) {
		fprintf(stdout, "%1.0f ", h_A[i]);
	}*/

	// Free host memory
	free(h_A); free(h_b); free(h_c);
	// Free GPU memory
	hipFree(d_A); hipFree(d_b); hipFree(d_c);

	// Shutdown
	status = cublasShutdown();
	if (status != HIPBLAS_STATUS_SUCCESS) {
		_error_handler("shutdown error (A)\n");
	}

	return EXIT_SUCCESS;
}
